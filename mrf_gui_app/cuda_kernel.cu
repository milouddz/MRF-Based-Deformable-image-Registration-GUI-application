// cuda_kernel.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void squareKernel(float* d_out, float* d_in, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_out[idx] = d_in[idx] * d_in[idx];
    }
}

extern "C" void squareArray(float* h_out, float* h_in, int size) {
    float* d_in;
    float* d_out;
    size_t bytes = size * sizeof(float);

    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);

    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    squareKernel<<<gridSize, blockSize>>>(d_out, d_in, size);

    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}
